
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>

static int ROW_A = 16384;
static int COL_A = 16384;
static int COL_B = 16384;

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_sec + (double)1e-6 * tv.tv_usec;
}

void mat_mul_seq(float *A, float *B, float *C,
                 int ROW_A, int COL_A, int COL_B);
void mat_mul_opencl(float *A, float *B, float *C,
                    int ROW_A, int COL_A, int COL_B);
void mat_mul_cuda(float *A, float *B, float *C,
                    int ROW_A, int COL_A, int COL_B);
void mat_mul_cuda_multi(float *A, float *B, float *C,
                    int ROW_A, int COL_A, int COL_B);
void mat_mul_cublas(float *A, float *B, float *C,
                    int ROW_A, int COL_A, int COL_B);
void verify(float *A, float *B, float *C,
            int ROW_A, int COL_A, int COL_B);

int main(int argc, char *argv[]) {
  if (argc < 2) {
    printf("Usage: %s <option>\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  int option = atoi(argv[1]);

  float *A = (float*)malloc(sizeof(float) * ROW_A * COL_A);
  float *B = (float*)malloc(sizeof(float) * COL_A * COL_B);
  float *C = (float*)malloc(sizeof(float) * ROW_A * COL_B);
  int i, j;

  for (i = 0; i < ROW_A; i++) {
    for (j = 0; j < COL_A; j++) {
      A[i * COL_A + j] = (float)(rand() % 1000) / 100.0f;
    }
  }
  for (i = 0; i < COL_A; i++) {
    for (j = 0; j < COL_B; j++) {
      B[i * COL_B + j] = (float)(rand() % 1000) / 100.0f;
    }
  }

  printf("Matrix Multiplication\n");
  printf("C[%lu X %lu] = A[%lu X %lu] X B[%lu X %lu]\n",
         ROW_A, COL_B, ROW_A, COL_A, COL_A, COL_B);

  double start_time = get_time();

  if (option == 0) {
    printf("Sequential version...\n");
    mat_mul_seq(A, B, C, ROW_A, COL_A, COL_B);
  } else if (option == 1) {
    printf("OpenCL version...\n");
    mat_mul_opencl(A, B, C, ROW_A, COL_A, COL_B);
  } else if (option == 2) {
    printf("CUDA version...\n");
    mat_mul_cuda(A, B, C, ROW_A, COL_A, COL_B);
  } else if (option == 3) {
    printf("CUDA multi device version...\n");
    mat_mul_cuda_multi(A, B, C, ROW_A, COL_A, COL_B);
  } else if (option == 4) {
    printf("cuBLAS version...\n");
    mat_mul_cublas(A, B, C, ROW_A, COL_A, COL_B);
  } else {
    printf("Invalid option!\n");
    exit(EXIT_FAILURE);
  }

  double end_time = get_time();
  printf("Elapsed time: %f sec\n", end_time - start_time);

  verify(A, B, C, ROW_A, COL_A, COL_B);

  free(A);
  free(B);
  free(C);
  return 0;
}

void verify(float *A, float *B, float *C,
            int ROW_A, int COL_A, int COL_B) {
  int i, j, k;
  float sum;

  for (i = 0; i < ROW_A; i+=97) {
    for (j = 0; j < COL_B; j+=89) {
      sum = 0.0f;
      for (k = 0; k < COL_A; k++) {
        sum += A[i * COL_A + k] * B[k * COL_B + j];
      }
      if (fabsf(C[i * COL_B + j] - sum) > 0.1) {
        printf("Verification failed! C[%d][%d]: %f vs. %f\n",
               i, j, C[i * COL_B + j], sum);
        return;
      }
    }
  }
  printf("Verification success!\n");
}
