
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void gpuVecAdd(float *A, float *B, float *C) {
  // TODO: write kernel code here
}

void init(float *V, int N) {
  for (int i = 0; i < N; i++) {
    V[i] = rand() % 100;
  }
}

void verify(float *A, float *B, float *C, int N) {
  for (int i = 0; i < 16384; i++) {
    if (A[i] + B[i] != C[i]) {
      printf("Verification failed! A[%d] = %d, B[%d] = %d, C[%d] = %d\n",
             i, A[i], i, B[i], i, C[i]);
      return;
    }
  }
  printf("Verification success!\n");
}

int main() {
  int N = 16384;

  float *A = (float*)malloc(sizeof(float) * N); 
  float *B = (float*)malloc(sizeof(float) * N); 
  float *C = (float*)malloc(sizeof(float) * N);

  init(A, N);
  init(B, N);

  // Memory objects of the device
  float *d_A, *d_B, *d_C;

  // TODO: allocate memory objects d_A, d_B, and d_C.

  // TODO: copy "A" to "d_A" (host to device).
  // TODO: copy "B" to "d_B" (host to device).

  // TODO: launch the kernel.

  // TODO: copy "d_C" to "C" (device to host).

  verify(A, B, C, N);

  // TODO: release d_A, d_B, and d_C.

  free(A);
  free(B);
  free(C);

  return 0;
}

